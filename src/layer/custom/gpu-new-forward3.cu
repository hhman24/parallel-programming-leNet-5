#include "hip/hip_runtime.h"
// Tran Tien Hoang - 20127424
#include "./gpu-new-forward.h"
#include <cmath>
#include <iostream>

#define TILE_WIDTH 16
#define KERNEL_SIZE_MAX 5 // Assume maximum kernel size is 5x5

__global__ void conv_forward_kernel(float *output, const float *input, const float *kernel,
                                    const int num_samples, const int output_channel, const int input_channel,
                                    const int height, const int width, const int kernel_size)
{
    const int height_out = height - kernel_size + 1;
    const int width_out = width - kernel_size + 1;

    int height_grid = ceil(1.0 * height_out / TILE_WIDTH);
    int width_grid = ceil(1.0 * width_out / TILE_WIDTH); 

    // Declare shared memory for input tile and kernel tile
    __shared__ float input_tile[TILE_WIDTH + KERNEL_SIZE_MAX - 1][TILE_WIDTH + KERNEL_SIZE_MAX - 1];
    __shared__ float kernel_tile[KERNEL_SIZE_MAX][KERNEL_SIZE_MAX];

    // Calculate block and thread indices
    int batch_idx = blockIdx.x;
    int output_feature_idx = blockIdx.y;
    int block_z = blockIdx.z;
    int block_row = block_z / width_grid;
    int block_col = block_z % width_grid;
    int row_idx = block_row * TILE_WIDTH + threadIdx.y;
    int col_idx = block_col * TILE_WIDTH + threadIdx.x;

    float accumulator = 0.0f;

    // Iterate over input channels
    for (int input_channel_idx = 0; input_channel_idx < input_channel; input_channel_idx++)
    {
        // Copy input tile to shared memory
        if (row_idx < height && col_idx < width)
        {
            input_tile[threadIdx.y][threadIdx.x] = input[(batch_idx * (input_channel * height * width)) +
                                                         (input_channel_idx * (height * width)) +
                                                         (row_idx * width) +
                                                         col_idx];
        }
        else
        {
            input_tile[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // Copy kernel tile to shared memory
        if (threadIdx.y < kernel_size && threadIdx.x < kernel_size)
        {
            kernel_tile[threadIdx.y][threadIdx.x] = kernel[(output_feature_idx * (input_channel * kernel_size * kernel_size)) +
                                                           (input_channel_idx * (kernel_size * kernel_size)) +
                                                           (threadIdx.y * kernel_size) +
                                                           threadIdx.x];
        }

        __syncthreads();

        // Convolution computation using shared memory
        for (int k_row = 0; k_row < kernel_size; k_row++)
        {
            for (int k_col = 0; k_col < kernel_size; k_col++)
            {
                accumulator += input_tile[threadIdx.y + k_row][threadIdx.x + k_col] * kernel_tile[k_row][k_col];
            }
        }

        __syncthreads();
    }

    // Store result in output
    if (row_idx < height_out && col_idx < width_out)
    {
        output[(batch_idx * (output_channel * height_out * width_out)) +
               (output_feature_idx * (height_out * width_out)) +
               (row_idx * width_out) +
               col_idx] = accumulator;
    }
}

__host__ void GPUInterface::conv_forward_gpu_full(float *output_data, const float *input_data, const float *weight_data,
                                                  const int num_samples, const int output_channel, const int input_channel,
                                                  const int height_in, const int width_in, const int kernel_height)
{
    const int height_out = height_in - kernel_height + 1;
    const int width_out = width_in - kernel_height + 1;

    // Allocate device memory
    float *device_input, *device_output, *device_weight;
    hipMalloc((void **)&device_input, num_samples * input_channel * height_in * width_in * sizeof(float));              // input features map is input_channel
    hipMalloc((void **)&device_output, num_samples * output_channel * height_out * width_out * sizeof(float));          // output feature map is output_channel
    hipMalloc((void **)&device_weight, output_channel * input_channel * kernel_height * kernel_height * sizeof(float)); // input_channel * output_channel filter Maps of size kernel_height * kernel_height

    // Copy input and mask data to device
    hipMemcpy(device_input, input_data, num_samples * input_channel * height_in * width_in * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_weight, weight_data, output_channel * input_channel * kernel_height * kernel_height * sizeof(float), hipMemcpyHostToDevice);

    // Set the kernel dimensions and call the kernel
    int height_grid = ceil(1.0 * height_out / TILE_WIDTH);
    int width_grid = ceil(1.0 * width_out / TILE_WIDTH);
    int Z = height_grid * width_grid;
    dim3 num_threads_per_block(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 num_blocks_in_grid(num_samples, output_channel, Z);

    // Launch the kernel
    conv_forward_kernel<<<num_blocks_in_grid, num_threads_per_block>>>(device_output, device_input, device_weight, num_samples, output_channel, input_channel, height_in, width_in, kernel_height);

    // Copy the output back to host
    hipMemcpy(output_data, device_output, num_samples * output_channel * height_out * width_out * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_weight);
}
