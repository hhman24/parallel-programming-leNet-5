#include "gpu-support.h"

char *GPU_Support::concatStr(const char *s1, const char *s2)
{
	char *result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
	strcpy(result, s1);
	strcat(result, s2);
	return result;
}

void GPU_Support::printDeviceInfo()
{
	hipDeviceProp_t devProv;
	CHECK(hipGetDeviceProperties(&devProv, 0));
	printf("**********GPU info**********\n");
	printf("Name: %s\n", devProv.name);
	printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
	printf("Num SMs: %d\n", devProv.multiProcessorCount);
	printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
	printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
	printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
	printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
	printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
	printf("****************************\n");
}